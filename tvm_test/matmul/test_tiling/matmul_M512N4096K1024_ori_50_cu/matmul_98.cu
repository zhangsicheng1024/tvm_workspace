
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[512];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[256];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 16; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 8) + (i_c_inner_init * 4)) + j_c_outer_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 8) + (i_c_inner_init * 4)) + j_c_outer_inner_init) + 128)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 8) + (i_c_inner_init * 4)) + j_c_outer_inner_init) + 256)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 8) + (i_c_inner_init * 4)) + j_c_outer_inner_init) + 384)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 32; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 32) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 6) * 262144) + (ax0_ax1_fused_outer_outer * 8192)) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 8; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 32) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 16384) + ((ax0_ax1_fused_outer_outer_1 >> 1) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((ax0_ax1_fused_outer_outer_1 & 1) * 32)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 16; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
          for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
            T_matmul_NN_local[(((i_c_outer_inner * 8) + (i_c_inner * 4)) + j_c_outer_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 8) + (i_c_inner * 4)) + j_c_outer_inner)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 128) + (i_c_outer_inner * 8)) + (i_c_inner * 4)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + j_c_outer_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 4)) + j_c_outer_inner) + 128)] = (T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 4)) + j_c_outer_inner) + 128)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 128) + (i_c_outer_inner * 8)) + (i_c_inner * 4)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + j_c_outer_inner) + 32)]));
            T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 4)) + j_c_outer_inner) + 256)] = (T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 4)) + j_c_outer_inner) + 256)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 128) + (i_c_outer_inner * 8)) + (i_c_inner * 4)) + k_outer_inner) + 512)] * kernel_shared[(((k_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + j_c_outer_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 4)) + j_c_outer_inner) + 384)] = (T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 4)) + j_c_outer_inner) + 384)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 128) + (i_c_outer_inner * 8)) + (i_c_inner * 4)) + k_outer_inner) + 512)] * kernel_shared[((((k_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + j_c_outer_inner) + 32)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 32; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 3) * 131072)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 3) * 131072)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_inner) + 32)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 128)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 3) * 131072)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_inner) + 524288)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 256)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 3) * 131072)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_inner) + 524320)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 384)];
    }
  }
}
